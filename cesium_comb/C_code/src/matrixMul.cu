#include "matrixMul.h"
#include <time.h>
#define SCALE(vector,num)\
{const hipDoubleComplex alpha = make_hipDoubleComplex(num,0.0);  \
  hipblasZscal(handle,N*N,&alpha,vector,1);\
}\

#define MUL(x,y,result)                               \
  {             \
  const hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);\
  const hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);\
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, x, N, y, N, &beta, result, N);\
  }             \

#define MUL1(x,y,result)                               \
  {             \
  hipMalloc((void**) &d_##result, mem_size);  \
  const hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);\
  const hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);\
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, x, N, y, N, &beta, d_##result, N);\
  hipMemcpy(result, d_##result, mem_size, hipMemcpyDeviceToHost) ;\
  hipFree(d_##result);                                         \
  }             \

#define COPY(from,to)  hipblasZcopy(handle,N*N,from,1,to,1)
// y = ax + y
#define AXPY(y,x,a)\
{\
  const hipDoubleComplex alpha = make_hipDoubleComplex(a,0.0);\
  hipblasZaxpy (handle,N*N, &alpha , x , 1 , y , 1);             \
}\

#define AXPY1(y,x,a)\
{\
  hipMalloc((void**) &d_##x, mem_size);\
  hipMemcpy(d_##x, x, mem_size, hipMemcpyHostToDevice) ;         \
  const hipDoubleComplex alpha = make_hipDoubleComplex(a,0.0);\
  hipblasZaxpy (handle,N*N, &alpha , d_##x , 1 , y , 1);             \
  hipFree(d_##x);  \
}\



void progressBar(unsigned int full,long avg_time)
{
  static unsigned int now = 1;
  long eta = (full - now)*avg_time;
  long day = eta/86400;
  eta = eta % 86400;
  long hour = eta/3600;
  eta = eta % 3600;
  long minute = eta/60;
  eta = eta % 60;    
  printf("%d%% \t average time: %ld,\t ETA:%ld day %ld hour %ld minute %ld seconds\r",now*100/full,avg_time,day,hour,minute,eta);
  fflush(stdout);
  now++;

}

void doubleMatrixPrint(double* data,unsigned int size)
{
  for (int i = 0; i < size; ++i)
    {
      printf("%d:\t %f \n",i,data[i]);
    }
}

hipDoubleComplex* complexMatrixCreate(double* real,double* imag,unsigned int size)
{
    hipDoubleComplex* A  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*size);
    for (int i = 0; i < size; ++i)
      {
        A[i] = make_hipDoubleComplex(real[i],imag[i]);
      }
    return A;
}

hipDoubleComplex* complexIdentityMatrix(unsigned int size)
{
  hipDoubleComplex* A  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*(size*size));
  for (int i = 0; i < size*size; ++i)
    {
      A[i] = make_hipDoubleComplex(0.0,0.0);
    }
  for (int i = 0; i < size; ++i)
    {
      A[i*size+i] = make_hipDoubleComplex(1.0,0.0);
    }
  return A;
}

void deviceVerify ()
{
  int devID;
  hipDeviceProp_t props;
  // get number of SMs on this GPU
  hipGetDevice(&devID);
  hipGetDeviceProperties(&props, devID);
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);
}

void complexTest()
{
  /*
    http://bccd-ng.cluster.earlham.edu/svn/bccd-ng/branches/lemanal-devel/trees/software/cluster/software/cuda/include/hip/hip_complex.h
    usage of cuda complex.
    hipCreal
    hipCimag
    hipCadd
    hipCsub
    hipCmul
    hipCdiv
    hipCabs
   */
  hipDoubleComplex* A  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*10);
  hipDoubleComplex* B  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*10);
  randomInit(A,10);
  randomInit(B,10);
  printMatrix(A,10);
  printMatrix(B,10);
  printDiff(A,B,3,3,100,1e-6);
  free(A);
  free(B);
}


//cpu reference
void computeGold(hipDoubleComplex* C, const hipDoubleComplex* A, const hipDoubleComplex* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
  for (unsigned int i = 0; i < hA; ++i)
    for (unsigned int j = 0; j < wB; ++j) {
      hipDoubleComplex sum = make_hipDoubleComplex(0.0,0.0);
      for (unsigned int k = 0; k < wA; ++k) {
        hipDoubleComplex a = A[i * wA + k];
        hipDoubleComplex b = B[k * wB + j];
        sum = hipCadd(sum,hipCmul(a,b));
      }
      C[i * wB + j] = sum;
    }
}


void inline checkError(hipblasStatus_t status, const char* msg)
{
  if(status != HIPBLAS_STATUS_SUCCESS){
    printf(msg);
    exit(-1);
  }
}

void randomInit(hipDoubleComplex* data, int size)
{
  // set seed for rand()
  srand(2006);
  for (int i = 0; i < size; ++i)
    {
      hipDoubleComplex tmp;
      tmp = make_hipDoubleComplex(rand()/(double)RAND_MAX,rand()/(double)RAND_MAX);
      data[i] = tmp;
    }

}

void returnMatrixPointer(hipDoubleComplex* data,char ri,int size, double* result)
{
  for (int i = 0; i < size; ++i)
    {
     if (ri == 'r')
       {
         result[i] = hipCreal(data[i]);
       }
     else
       {
         result[i] = hipCimag(data[i]);
       }
    }
}

void printMatrix(hipDoubleComplex* data, int size)
{
    for (int i = 0; i < size; ++i)
    {
      printf("%d:  %f + %f j \n",i,hipCreal(data[i]),hipCimag(data[i]));
    }
    printf("\n \n");

}

void printDiff(hipDoubleComplex *data1, hipDoubleComplex *data2, int width, int height, int iListLength, float fListTol)
{
  printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
  int i,j,k;
  int error_count=0;
  for (j = 0; j < height; j++)
    {
      if (error_count < iListLength)
        {
          printf("\n  Row %d:\n", j);
        }
      for (i = 0; i < width; i++)
        {
          k = j * width + i;
          float fDiff = hipCabs(hipCsub(data1[k],data2[k]));
          if (fDiff > fListTol)
            {
              if (error_count < iListLength)
                {
                  printf("    Loc(%d,%d)\tCPU=%.5f+%.5f j\tGPU=%.5f+%.5f j\tDiff=%.6f\n", i, j, hipCreal(data1[k]),hipCimag(data1[k]),hipCreal(data2[k]),hipCimag(data2[k]), fDiff);
                }
              error_count++;
            }
        }
    }
  printf(" \n  Total Errors = %d\n\n", error_count);
}

void solve(double* Hsr,double* Hsi, double* Her,double* Hei, double* E_arr,double dt,int N, int finestep,double* resultr,double*resulti)
{
  printf ("\nstart gpu solving...\n");
  printf ("matrix size:%d\n",N*N);
  printf ("step size:%d\n\n",finestep/6);

  unsigned int mem_size = sizeof(hipDoubleComplex) * (N*N);

  // allocate device memory
  hipDoubleComplex* d_Hs, *d_He, *d_result,* d_result_tmp, *d_k1, *d_k2, *d_k3, *d_k4, *d_k5, *d_k6, *d_tmp,*d_tmp2, *d_I;
  hipMalloc((void**) &d_Hs, mem_size);
  hipMalloc((void**) &d_He, mem_size);
  hipMalloc((void**) &d_result, mem_size);
  /* hipMalloc((void**) &d_k1, mem_size); */
  /* hipMalloc((void**) &d_k3, mem_size); */
  /* hipMalloc((void**) &d_k4, mem_size); */
  /* hipMalloc((void**) &d_k5, mem_size); */
  /* hipMalloc((void**) &d_k6, mem_size); */
  hipMalloc((void**) &d_tmp, mem_size);
  hipMalloc((void**) &d_tmp2, mem_size);
  hipMalloc((void**) &d_I, mem_size);

  hipDoubleComplex* k1 = (hipDoubleComplex*) malloc(mem_size);
  hipDoubleComplex* k2 = (hipDoubleComplex*) malloc(mem_size);
  hipDoubleComplex* k3 = (hipDoubleComplex*) malloc(mem_size);
  hipDoubleComplex* k4 = (hipDoubleComplex*) malloc(mem_size);
  hipDoubleComplex* k5 = (hipDoubleComplex*) malloc(mem_size);
  hipDoubleComplex* k6 = (hipDoubleComplex*) malloc(mem_size);

  hipDoubleComplex* Hs = complexMatrixCreate(Hsr,Hsi,N*N);
  hipDoubleComplex* He = complexMatrixCreate(Her,Hei,N*N);
  hipDoubleComplex* I = complexIdentityMatrix(N);
  hipDoubleComplex* result = complexIdentityMatrix(N);

  printf ("copy data to device...\n");
  // copy host memory to device
  hipMemcpy(d_I, I, mem_size, hipMemcpyHostToDevice) ;
  hipMemcpy(d_result, result, mem_size, hipMemcpyHostToDevice) ;
  hipMemcpy(d_Hs, Hs, mem_size, hipMemcpyHostToDevice) ;
  hipMemcpy(d_He, He, mem_size, hipMemcpyHostToDevice) ;


  printf ("initialize cublas\n");
  //cublas_v2
  hipblasHandle_t handle;
  checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");
  int counter = 0;
  long avg_time = 0;
  time_t seconds;

  for (int i = 0; i < finestep; i=i+6)
    {
      seconds = time (NULL);
      progressBar(finestep/6,avg_time);
      //printf ("%d\n",i);
      /* k1 */
      hipMalloc((void**) &d_k1, mem_size);
      COPY(d_He,d_k1);
      SCALE(d_k1,E_arr[i]);
      AXPY(d_k1,d_Hs,1.0);
      hipMemcpy(k1, d_k1, mem_size, hipMemcpyDeviceToHost) ;
      hipFree(d_k1);
      /* tmp */
      COPY(d_He,d_tmp);
      SCALE(d_tmp,E_arr[i+1]);
      AXPY(d_tmp,d_Hs,1.0);
      /* tmp2 */
      COPY(d_I,d_tmp2);

      AXPY1(d_tmp2,k1,0.25*dt);

      /* k2 */
      MUL1(d_tmp,d_tmp2,k2);
      /* tmp */
      COPY(d_He,d_tmp);
      SCALE(d_tmp,E_arr[i+2]);
      AXPY(d_tmp,d_Hs,1.0);
      /* tmp2 */
      COPY(d_I,d_tmp2);
      AXPY1(d_tmp2,k1,3.0/32.0*dt);
      AXPY1(d_tmp2,k2,9.0/32.0*dt);

      /* k3 */
      MUL1(d_tmp,d_tmp2,k3);
      /* tmp */
      COPY(d_He,d_tmp);
      SCALE(d_tmp,E_arr[i+3]);
      AXPY(d_tmp,d_Hs,1.0);
      /* tmp2 */
      COPY(d_I,d_tmp2);
      AXPY1(d_tmp2,k1,1932.0/2197.0*dt);
      AXPY1(d_tmp2,k2,-1.0*7200.0/2197.0*dt);
      AXPY1(d_tmp2,k3,7296.0/2197.0*dt);
      /* k4 */
      MUL1(d_tmp,d_tmp2,k4);
      /* tmp */
      COPY(d_He,d_tmp);
      SCALE(d_tmp,E_arr[i+4]);
      AXPY(d_tmp,d_Hs,1.0);
      /* tmp2 */
      COPY(d_I,d_tmp2);
      AXPY1(d_tmp2,k1,439.0/216.0*dt);
      AXPY1(d_tmp2,k2,-1.0*8.0*dt);
      AXPY1(d_tmp2,k3,3680.0/513.0*dt);
      AXPY1(d_tmp2,k4,-1.0*845.0/4104.0*dt);
      /* k5 */
      MUL1(d_tmp,d_tmp2,k5);

      /* tmp */
      COPY(d_He,d_tmp);
      SCALE(d_tmp,E_arr[i+5]);
      AXPY(d_tmp,d_Hs,1.0);
      /* tmp2 */
      COPY(d_I,d_tmp2);
      AXPY1(d_tmp2,k1,-1.0*8.0/27.0*dt);
      AXPY1(d_tmp2,k2,-1.0*2.0*dt);
      AXPY1(d_tmp2,k3,-1.0*3544.0/2565.0*dt);
      AXPY1(d_tmp2,k4,1859.0/4104.0*dt);
      AXPY1(d_tmp2,k5,-1.0*11.0/40.0*dt);
      hipFree(d_k2);
      /* k6 */
      MUL1(d_tmp,d_tmp2,k6);
      /* tmp */
      COPY(d_I,d_tmp);
      AXPY1(d_tmp,k1,16.0/135.0*dt);
      AXPY1(d_tmp,k3,6656.0/12825.0*dt);
      AXPY1(d_tmp,k4,28561.0/56430.0*dt);
      AXPY1(d_tmp,k5,-1.0*9.0/50.0*dt);
      AXPY1(d_tmp,k6,2.0/55.0*dt);
      /* result */
      hipMalloc((void**) &d_result_tmp, mem_size);
      MUL(d_tmp,d_result,d_result_tmp);
      COPY(d_result_tmp,d_result);
      hipFree(d_result_tmp);
      seconds = time (NULL) - seconds;

      avg_time = (avg_time*counter + seconds)/(counter+1);
      counter++;
    }

  //retrieve result
  hipMemcpy(result, d_result, mem_size, hipMemcpyDeviceToHost) ;
  checkError(hipblasDestroy(handle), "hipblasDestroy() error!\n");
  returnMatrixPointer(result,'r',N*N,resultr);
  returnMatrixPointer(result,'i',N*N,resulti);

  //free memory
  hipFree(d_Hs);
  hipFree(d_He);
  hipFree(d_result);
  hipFree(d_result_tmp);
  hipFree(d_k1);
  hipFree(d_k2);
  hipFree(d_k3);
  hipFree(d_k4);
  hipFree(d_k5);
  hipFree(d_k6);
  hipFree(d_tmp);
  hipFree(d_tmp2);
}

void runTest()
{
  unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
    uiWA = 2;
    uiHA = 2;
    uiWB = 2;
    uiHB = 2;
    uiWC = 2;
    uiHC = 2;
  printf("\nUsing Matrix Sizes: A(%u x %u), B(%u x %u), C(%u x %u)\n\n",
         uiWA, uiHA, uiWB, uiHB, uiWC, uiHC);

  // allocate host memory for matrices A and B
  unsigned int size_A = uiWA * uiHA;
  unsigned int mem_size_A = sizeof(hipDoubleComplex) * size_A;
  hipDoubleComplex* h_A = (hipDoubleComplex*)malloc(mem_size_A);
  unsigned int size_B = uiWB * uiHB;
  unsigned int mem_size_B = sizeof(hipDoubleComplex) * size_B;
  hipDoubleComplex* h_B = (hipDoubleComplex*)malloc(mem_size_B);

  // initialize host memory
  randomInit(h_A, size_A);
  randomInit(h_B, size_B);
  printMatrix(h_A, size_A);
  printMatrix(h_B, size_B);

  // allocate device memory
  hipDoubleComplex* d_A, *d_B, *d_C;
  unsigned int size_C = uiWC * uiHC;
  unsigned int mem_size_C = sizeof(hipDoubleComplex) * size_C;

  // allocate host memory for the result
  hipDoubleComplex* h_C      = (hipDoubleComplex*) malloc(mem_size_C);
  hipDoubleComplex* h_CUBLAS = (hipDoubleComplex*) malloc(mem_size_C);

  hipMalloc((void**) &d_A, mem_size_A);
  hipMalloc((void**) &d_B, mem_size_B);

  // copy host memory to device
  hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) ;
  hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) ;

  hipMalloc((void**) &d_C, mem_size_C);


  //cublas_v2
  hipblasHandle_t handle;
  checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");
  const hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
  const hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);

  //note cublas is column primary!
  //need to transpose the order
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, uiWB, uiHA, uiWA, &alpha, d_B, uiWB, d_A, uiWA, &beta, d_C, uiWA);

  // copy result from device to host
  hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost) ;
  checkError(hipblasDestroy(handle), "hipblasDestroy() error!\n");

  // compute reference solution
  hipDoubleComplex* reference = (hipDoubleComplex*)malloc(mem_size_C);
  computeGold(reference, h_A, h_B, uiHA, uiWA, uiWB);

  // check result (CUBLAS)
  printf("Comparing CUBLAS & Host results\n");
  printDiff(reference, h_CUBLAS, uiWC, uiHC, 100, 1.0e-5f);
  printMatrix(h_CUBLAS, size_C);

  // clean up memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(reference);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

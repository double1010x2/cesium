#include "matrixMul.h"
#define SCALE(vector,num)\
{const hipDoubleComplex alpha = make_hipDoubleComplex(num,0.0);  \
  hipblasZscal(handle,N*N,&alpha,vector,1);\
}\

#define MUL(x,y,result)                               \
  {             \
  const hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);\
  const hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);\
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, x, N, y, N, &beta, result, N);\
  }             \


#define COPY(from,to)  hipblasZcopy(handle,N*N,from,1,to,1)
// y = ax + y
#define AXPY(y,x,a)\
{\
  const hipDoubleComplex alpha = make_hipDoubleComplex(a,0.0);\
  hipblasZaxpy (handle,N*N, &alpha , x , 1 , y , 1);             \
}\



void progressBar(unsigned int full)
{
  static unsigned int now = 1;
  printf("%d\% \r",now*100/full);
  fflush(stdout);
  now++;

}

void doubleMatrixPrint(double* data,unsigned int size)
{
  for (int i = 0; i < size; ++i)
    {
      printf("%d:\t %f \n",i,data[i]);
    }
}

hipDoubleComplex* complexMatrixCreate(double* real,double* imag,unsigned int size)
{
    hipDoubleComplex* A  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*size);
    for (int i = 0; i < size; ++i)
      {
        A[i] = make_hipDoubleComplex(real[i],imag[i]);
      }
    return A;
}

hipDoubleComplex* complexIdentityMatrix(unsigned int size)
{
  hipDoubleComplex* A  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*(size*size));
  for (int i = 0; i < size*size; ++i)
    {
      A[i] = make_hipDoubleComplex(0.0,0.0);
    }
  for (int i = 0; i < size; ++i)
    {
      A[i*size+i] = make_hipDoubleComplex(1.0,0.0);
    }
  return A;
}

void deviceVerify ()
{
  int devID;
  hipDeviceProp_t props;
  // get number of SMs on this GPU
  hipGetDevice(&devID);
  hipGetDeviceProperties(&props, devID);
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);
}

void complexTest()
{
  /*
    http://bccd-ng.cluster.earlham.edu/svn/bccd-ng/branches/lemanal-devel/trees/software/cluster/software/cuda/include/hip/hip_complex.h
    usage of cuda complex.
    hipCreal
    hipCimag
    hipCadd
    hipCsub
    hipCmul
    hipCdiv
    hipCabs
   */
  hipDoubleComplex* A  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*10);
  hipDoubleComplex* B  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*10);
  randomInit(A,10);
  randomInit(B,10);
  printMatrix(A,10);
  printMatrix(B,10);
  printDiff(A,B,3,3,100,1e-6);
  free(A);
  free(B);
}


//cpu reference
void computeGold(hipDoubleComplex* C, const hipDoubleComplex* A, const hipDoubleComplex* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
  for (unsigned int i = 0; i < hA; ++i)
    for (unsigned int j = 0; j < wB; ++j) {
      hipDoubleComplex sum = make_hipDoubleComplex(0.0,0.0);
      for (unsigned int k = 0; k < wA; ++k) {
        hipDoubleComplex a = A[i * wA + k];
        hipDoubleComplex b = B[k * wB + j];
        sum = hipCadd(sum,hipCmul(a,b));
      }
      C[i * wB + j] = sum;
    }
}


void inline checkError(hipblasStatus_t status, const char* msg)
{
  if(status != HIPBLAS_STATUS_SUCCESS){
    printf(msg);
    exit(-1);
  }
}

void randomInit(hipDoubleComplex* data, int size)
{
  // set seed for rand()
  srand(2006);
  for (int i = 0; i < size; ++i)
    {
      hipDoubleComplex tmp;
      tmp = make_hipDoubleComplex(rand()/(double)RAND_MAX,rand()/(double)RAND_MAX);
      data[i] = tmp;
    }

}

void returnMatrixPointer(hipDoubleComplex* data,char ri,int size, double* result)
{
  for (int i = 0; i < size; ++i)
    {
     if (ri == 'r')
       {
         result[i] = hipCreal(data[i]);
       }
     else
       {
         result[i] = hipCimag(data[i]);
       }
    }
}

void printMatrix(hipDoubleComplex* data, int size)
{
    for (int i = 0; i < size; ++i)
    {
      printf("%d:  %f + %f j \n",i,hipCreal(data[i]),hipCimag(data[i]));
    }
    printf("\n \n");

}

void printDiff(hipDoubleComplex *data1, hipDoubleComplex *data2, int width, int height, int iListLength, float fListTol)
{
  printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
  int i,j,k;
  int error_count=0;
  for (j = 0; j < height; j++)
    {
      if (error_count < iListLength)
        {
          printf("\n  Row %d:\n", j);
        }
      for (i = 0; i < width; i++)
        {
          k = j * width + i;
          float fDiff = hipCabs(hipCsub(data1[k],data2[k]));
          if (fDiff > fListTol)
            {
              if (error_count < iListLength)
                {
                  printf("    Loc(%d,%d)\tCPU=%.5f+%.5f j\tGPU=%.5f+%.5f j\tDiff=%.6f\n", i, j, hipCreal(data1[k]),hipCimag(data1[k]),hipCreal(data2[k]),hipCimag(data2[k]), fDiff);
                }
              error_count++;
            }
        }
    }
  printf(" \n  Total Errors = %d\n\n", error_count);
}

void solve(hipDoubleComplex* Hs, hipDoubleComplex* He,hipDoubleComplex* result, double* E_arr,double dt,int N, int finestep)
{
  printf ("\nstart gpu solving...\n");
  printf ("matrix size:%d\n",N*N);
  printf ("step size:%d\n\n",finestep/6);

  unsigned int mem_size = sizeof(hipDoubleComplex) * (N*N);

  // allocate device memory
  hipDoubleComplex* d_Hs, *d_He, *d_result,* d_result_tmp, *d_k1, *d_k2, *d_k3, *d_k4, *d_k5, *d_k6, *d_tmp,*d_tmp2, *d_I;
  hipMalloc((void**) &d_Hs, mem_size);
  hipMalloc((void**) &d_He, mem_size);
  hipMalloc((void**) &d_result, mem_size);
  hipMalloc((void**) &d_result_tmp, mem_size);  
  hipMalloc((void**) &d_k1, mem_size);
  hipMalloc((void**) &d_k2, mem_size);
  hipMalloc((void**) &d_k3, mem_size);
  hipMalloc((void**) &d_k4, mem_size);
  hipMalloc((void**) &d_k5, mem_size);
  hipMalloc((void**) &d_k6, mem_size);
  hipMalloc((void**) &d_tmp, mem_size);
  hipMalloc((void**) &d_tmp2, mem_size);
  hipMalloc((void**) &d_I, mem_size);

  hipDoubleComplex* I = complexIdentityMatrix(N);

  // copy host memory to device
  hipMemcpy(d_Hs, Hs, mem_size, hipMemcpyHostToDevice) ;
  hipMemcpy(d_He, He, mem_size, hipMemcpyHostToDevice) ;
  hipMemcpy(d_result, result, mem_size, hipMemcpyHostToDevice) ;
  hipMemcpy(d_I, I, mem_size, hipMemcpyHostToDevice) ;

  //cublas_v2
  hipblasHandle_t handle;
  checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");

  //test: scale result  y = ax + y

  /* COPY(d_result,d_tmp); */
  /* AXPY(d_tmp,d_result,2.0); */
  /* MUL(d_result,d_Hs,d_tmp); */
  /* COPY(d_tmp,d_result); */

  for (int i = 0; i < finestep; i=i+6)
    {
      progressBar(finestep/6);
      /* k1 */
      COPY(d_He,d_k1);
      SCALE(d_k1,E_arr[i]);
      AXPY(d_k1,d_Hs,1.0);
      /* tmp */
      COPY(d_He,d_tmp);
      SCALE(d_tmp,E_arr[i+1]);
      AXPY(d_tmp,d_Hs,1.0);
      /* tmp2 */
      COPY(d_I,d_tmp2);
      AXPY(d_tmp2,d_k1,0.25*dt);
      /* k2 */
      MUL(d_tmp,d_tmp2,d_k2);
      /* tmp */
      COPY(d_He,d_tmp);
      SCALE(d_tmp,E_arr[i+2]);
      AXPY(d_tmp,d_Hs,1.0);
      /* tmp2 */
      COPY(d_I,d_tmp2);
      AXPY(d_tmp2,d_k1,3.0/32.0*dt);
      AXPY(d_tmp2,d_k2,9.0/32.0*dt);
      /* k3 */
      MUL(d_tmp,d_tmp2,d_k3);
      /* tmp */
      COPY(d_He,d_tmp);
      SCALE(d_tmp,E_arr[i+3]);
      AXPY(d_tmp,d_Hs,1.0);
      /* tmp2 */
      COPY(d_I,d_tmp2);
      AXPY(d_tmp2,d_k1,1932.0/2197.0*dt);
      AXPY(d_tmp2,d_k2,-1.0*7200.0/2197.0*dt);
      AXPY(d_tmp2,d_k3,7296.0/2197.0*dt);
      /* k4 */
      MUL(d_tmp,d_tmp2,d_k4);
      /* tmp */
      COPY(d_He,d_tmp);
      SCALE(d_tmp,E_arr[i+4]);
      AXPY(d_tmp,d_Hs,1.0);
      /* tmp2 */
      COPY(d_I,d_tmp2);
      AXPY(d_tmp2,d_k1,439.0/216.0*dt);
      AXPY(d_tmp2,d_k2,-1.0*8.0*dt);
      AXPY(d_tmp2,d_k3,3680.0/513.0*dt);
      AXPY(d_tmp2,d_k4,-1.0*845.0/4104.0*dt);
      /* k5 */
      MUL(d_tmp,d_tmp2,d_k5);

      /* tmp */
      COPY(d_He,d_tmp);
      SCALE(d_tmp,E_arr[i+5]);
      AXPY(d_tmp,d_Hs,1.0);
      /* tmp2 */
      COPY(d_I,d_tmp2);
      AXPY(d_tmp2,d_k1,-1.0*8.0/27.0*dt);
      AXPY(d_tmp2,d_k2,-1.0*2.0*dt);
      AXPY(d_tmp2,d_k3,-1.0*3544.0/2565.0*dt);
      AXPY(d_tmp2,d_k4,1859.0/4104.0*dt);
      AXPY(d_tmp2,d_k5,-1.0*11.0/40.0*dt);      
      /* k6 */
      MUL(d_tmp,d_tmp2,d_k6);
      /* tmp */
      COPY(d_I,d_tmp);
      AXPY(d_tmp,d_k1,16.0/135.0*dt);
      AXPY(d_tmp,d_k3,6656.0/12825.0*dt);
      AXPY(d_tmp,d_k4,28561.0/56430.0*dt);
      AXPY(d_tmp,d_k5,-1.0*9.0/50.0*dt);
      AXPY(d_tmp,d_k6,2.0/55.0*dt);
      /* result */
      MUL(d_tmp,d_result,d_result_tmp);
      COPY(d_result_tmp,d_result);
    }

  //retrieve result
  hipMemcpy(result, d_result, mem_size, hipMemcpyDeviceToHost) ;
  checkError(hipblasDestroy(handle), "hipblasDestroy() error!\n");
  //free memory
  hipFree(d_Hs);
  hipFree(d_He);
  hipFree(d_result);
  hipFree(d_k1);
  hipFree(d_k2);
  hipFree(d_k3);
  hipFree(d_k4);
  hipFree(d_k5);
  hipFree(d_k6);
}

void runTest()
{
  unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
    uiWA = 2;
    uiHA = 2;
    uiWB = 2;
    uiHB = 2;
    uiWC = 2;
    uiHC = 2;
  printf("\nUsing Matrix Sizes: A(%u x %u), B(%u x %u), C(%u x %u)\n\n",
         uiWA, uiHA, uiWB, uiHB, uiWC, uiHC);

  // allocate host memory for matrices A and B
  unsigned int size_A = uiWA * uiHA;
  unsigned int mem_size_A = sizeof(hipDoubleComplex) * size_A;
  hipDoubleComplex* h_A = (hipDoubleComplex*)malloc(mem_size_A);
  unsigned int size_B = uiWB * uiHB;
  unsigned int mem_size_B = sizeof(hipDoubleComplex) * size_B;
  hipDoubleComplex* h_B = (hipDoubleComplex*)malloc(mem_size_B);

  // initialize host memory
  randomInit(h_A, size_A);
  randomInit(h_B, size_B);
  printMatrix(h_A, size_A);
  printMatrix(h_B, size_B);

  // allocate device memory
  hipDoubleComplex* d_A, *d_B, *d_C;
  unsigned int size_C = uiWC * uiHC;
  unsigned int mem_size_C = sizeof(hipDoubleComplex) * size_C;

  // allocate host memory for the result
  hipDoubleComplex* h_C      = (hipDoubleComplex*) malloc(mem_size_C);
  hipDoubleComplex* h_CUBLAS = (hipDoubleComplex*) malloc(mem_size_C);

  hipMalloc((void**) &d_A, mem_size_A);
  hipMalloc((void**) &d_B, mem_size_B);

  // copy host memory to device
  hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) ;
  hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) ;

  hipMalloc((void**) &d_C, mem_size_C);


  //cublas_v2
  hipblasHandle_t handle;
  checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");
  const hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
  const hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);

  //note cublas is column primary!
  //need to transpose the order
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, uiWB, uiHA, uiWA, &alpha, d_B, uiWB, d_A, uiWA, &beta, d_C, uiWA);

  // copy result from device to host
  hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost) ;
  checkError(hipblasDestroy(handle), "hipblasDestroy() error!\n");

  // compute reference solution
  hipDoubleComplex* reference = (hipDoubleComplex*)malloc(mem_size_C);
  computeGold(reference, h_A, h_B, uiHA, uiWA, uiWB);

  // check result (CUBLAS)
  printf("Comparing CUBLAS & Host results\n");
  printDiff(reference, h_CUBLAS, uiWC, uiHC, 100, 1.0e-5f);
  printMatrix(h_CUBLAS, size_C);

  // clean up memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(reference);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

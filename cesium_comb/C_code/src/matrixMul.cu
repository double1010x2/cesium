#include "matrixMul.h"

void progressBar(unsigned int full)
{
  static unsigned int now = 1;
  printf("%d\% \r",now*100/full);
  fflush(stdout);
  now++;
  
}

void doubleMatrixPrint(double* data,unsigned int size)
{
  for (int i = 0; i < size; ++i)
    {
      printf("%d:\t %f \n",i,data[i]);
    }
}

hipDoubleComplex* complexMatrixCreate(double* real,double* imag,unsigned int size)
{
    hipDoubleComplex* A  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*size);
    for (int i = 0; i < size; ++i)
      {
        A[i] = make_hipDoubleComplex(real[i],imag[i]);
      }
    return A;
}

hipDoubleComplex* complexIdentityMatrix(unsigned int size)
{
  hipDoubleComplex* A  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*(size*size));
  for (int i = 0; i < size*size; ++i)
    {
      A[i] = make_hipDoubleComplex(0.0,0.0);
    }
  for (int i = 0; i < size; ++i)
    {
      A[i*size+i] = make_hipDoubleComplex(1.0,0.0);
    }
  return A;
}

void deviceVerify ()
{
  int devID;
  hipDeviceProp_t props;
  // get number of SMs on this GPU
  hipGetDevice(&devID);
  hipGetDeviceProperties(&props, devID);
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);
}

void complexTest()
{
  /*
    http://bccd-ng.cluster.earlham.edu/svn/bccd-ng/branches/lemanal-devel/trees/software/cluster/software/cuda/include/hip/hip_complex.h    
    usage of cuda complex.
    hipCreal
    hipCimag
    hipCadd
    hipCsub
    hipCmul
    hipCdiv
    hipCabs
   */
  hipDoubleComplex* A  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*10);
  hipDoubleComplex* B  = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex)*10);  
  randomInit(A,10);
  randomInit(B,10);  
  printMatrix(A,10);
  printMatrix(B,10);  
  printDiff(A,B,3,3,100,1e-6);
  free(A);
  free(B);
}


//cpu reference
void computeGold(hipDoubleComplex* C, const hipDoubleComplex* A, const hipDoubleComplex* B, unsigned int hA, unsigned int wA, unsigned int wB)  
{
  for (unsigned int i = 0; i < hA; ++i)
    for (unsigned int j = 0; j < wB; ++j) {
      hipDoubleComplex sum = make_hipDoubleComplex(0.0,0.0);
      for (unsigned int k = 0; k < wA; ++k) {
        hipDoubleComplex a = A[i * wA + k];
        hipDoubleComplex b = B[k * wB + j];
        sum = hipCadd(sum,hipCmul(a,b));
      }
      C[i * wB + j] = sum;
    }
}


void inline checkError(hipblasStatus_t status, const char* msg)
{
  if(status != HIPBLAS_STATUS_SUCCESS){
    printf(msg);
    exit(-1);
  }
}

void randomInit(hipDoubleComplex* data, int size)
{
  // set seed for rand()
  srand(2006);  
  for (int i = 0; i < size; ++i)
    {
      hipDoubleComplex tmp;
      tmp = make_hipDoubleComplex(rand()/(double)RAND_MAX,rand()/(double)RAND_MAX);
      data[i] = tmp;
    }
  
}

void printMatrix(hipDoubleComplex* data, int size)
{
    for (int i = 0; i < size; ++i)
    {
      printf("%d:  %f + %f j \n",i,hipCreal(data[i]),hipCimag(data[i]));
    }
    printf("\n \n");
    
}

void printDiff(hipDoubleComplex *data1, hipDoubleComplex *data2, int width, int height, int iListLength, float fListTol)
{
  printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
  int i,j,k;
  int error_count=0;
  for (j = 0; j < height; j++) 
    {
      if (error_count < iListLength)
        {
          printf("\n  Row %d:\n", j);
        }
      for (i = 0; i < width; i++) 
        {
          k = j * width + i;
          float fDiff = hipCabs(hipCsub(data1[k],data2[k]));
          if (fDiff > fListTol) 
            {                
              if (error_count < iListLength)
                {
                  printf("    Loc(%d,%d)\tCPU=%.5f+%.5f j\tGPU=%.5f+%.5f j\tDiff=%.6f\n", i, j, hipCreal(data1[k]),hipCimag(data1[k]),hipCreal(data2[k]),hipCimag(data2[k]), fDiff);
                }
              error_count++;
            }
        }
    }
  printf(" \n  Total Errors = %d\n\n", error_count);
}


void runTest()
{
  unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
    uiWA = 2;
    uiHA = 2;
    uiWB = 2;
    uiHB = 2;
    uiWC = 2;
    uiHC = 2;
  printf("\nUsing Matrix Sizes: A(%u x %u), B(%u x %u), C(%u x %u)\n\n",
         uiWA, uiHA, uiWB, uiHB, uiWC, uiHC);

  // allocate host memory for matrices A and B
  unsigned int size_A = uiWA * uiHA;
  unsigned int mem_size_A = sizeof(hipDoubleComplex) * size_A;
  hipDoubleComplex* h_A = (hipDoubleComplex*)malloc(mem_size_A);
  unsigned int size_B = uiWB * uiHB;
  unsigned int mem_size_B = sizeof(hipDoubleComplex) * size_B;
  hipDoubleComplex* h_B = (hipDoubleComplex*)malloc(mem_size_B);

  // initialize host memory
  randomInit(h_A, size_A);
  randomInit(h_B, size_B);
  printMatrix(h_A, size_A);
  printMatrix(h_B, size_B);
  
  // allocate device memory
  hipDoubleComplex* d_A, *d_B, *d_C;
  unsigned int size_C = uiWC * uiHC;
  unsigned int mem_size_C = sizeof(hipDoubleComplex) * size_C;

  // allocate host memory for the result
  hipDoubleComplex* h_C      = (hipDoubleComplex*) malloc(mem_size_C);
  hipDoubleComplex* h_CUBLAS = (hipDoubleComplex*) malloc(mem_size_C);

  hipMalloc((void**) &d_A, mem_size_A);
  hipMalloc((void**) &d_B, mem_size_B);

  // copy host memory to device
  hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) ;
  hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) ;
    
  hipMalloc((void**) &d_C, mem_size_C);

    
  //cublas_v2
  hipblasHandle_t handle;
  checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");
  const hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
  const hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);

  //note cublas is column primary!
  //need to transpose the order
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, uiWB, uiHA, uiWA, &alpha, d_B, uiWB, d_A, uiWA, &beta, d_C, uiWA);
  
  // copy result from device to host
  hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost) ;
  checkError(hipblasDestroy(handle), "hipblasDestroy() error!\n");

  // compute reference solution
  hipDoubleComplex* reference = (hipDoubleComplex*)malloc(mem_size_C);
  computeGold(reference, h_A, h_B, uiHA, uiWA, uiWB);

  // check result (CUBLAS)
  printf("Comparing CUBLAS & Host results\n");
  printDiff(reference, h_CUBLAS, uiWC, uiHC, 100, 1.0e-5f);
  printMatrix(h_CUBLAS, size_C);
  
  // clean up memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(reference);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

